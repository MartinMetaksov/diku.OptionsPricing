
// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#include "../common/Args.hpp"
#include "Version1.cuh"

using namespace std;
using namespace trinom;

void computeAllOptions(const Args &args)
{
    if (args.test)
    {
        cout << "Cuda multiple options per thread block version " << args.version << endl;
    }
    
    // Read options and yield curve.
    Options options(args.options);
    Yield yield(args.yield);

    hipFree(0);
    auto time_begin = steady_clock::now();

    vector<real> results;
    results.resize(options.N);

    switch (args.version)
    {
        case 1:
            cuda::computeOptionsNaive(options, yield, results, 64, args.test);
            break;
    }

    auto time_end = steady_clock::now();

    if (!args.test)
    {
        Arrays::write_array(cout, results);
    }
    else
    {
        cout << "Total execution time " << duration_cast<milliseconds>(time_end - time_begin).count() << " ms" << endl;
    }
}

int main(int argc, char *argv[])
{
    Args args(argc, argv);

    computeAllOptions(args);

    return 0;
}

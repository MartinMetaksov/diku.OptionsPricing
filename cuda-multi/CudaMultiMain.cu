
// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#include "Version1.cuh"
#include "Version2.cuh"
#include "Version3.cuh"
#include "../common/Args.hpp"

using namespace std;
using namespace trinom;

cuda::CudaRuntime run(const Options &options, const Yield &yield, vector<real> &results, const Args &args)
{
    switch (args.version)
    {
        case 1:
        {
            cuda::multi::KernelRunNaive kernelRun;
            kernelRun.run(options, yield, results, args.blockSize, args.sort, args.test);
            return kernelRun.runtime;
        }
        case 2:
        {
            cuda::multi::KernelRunCoalesced kernelRun;
            kernelRun.run(options, yield, results, args.blockSize, args.sort, args.test);
            return kernelRun.runtime;
        }
        case 3:
        {
            cuda::multi::KernelRunCoalescedBlock kernelRun;
            kernelRun.run(options, yield, results, args.blockSize, args.sort, args.test);
            return kernelRun.runtime;
        }
    }
    return cuda::CudaRuntime();
}

void computeAllOptions(const Args &args)
{
    if (args.test)
    {
        cout << "Cuda multiple options per thread block version " << args.version << endl;
    }
    
    // Read options and yield curve.
    Options options(args.options);
    Yield yield(args.yield);

    hipFree(0);

    if (args.runs > 0)
    {
        cout << "Performing " << args.runs << " runs..." << endl;
        cuda::CudaRuntime best;
        for (auto i = 0; i < args.runs; ++i)
        {
            vector<real> results;
            results.resize(options.N);
            auto runtime = run(options, yield, results, args);
            if (runtime < best)
            {
                best = runtime;
            }
        }
        cout << "Best times: kernel " << best.KernelRuntime << " microsec, total " << best.TotalRuntime << " microsec." << endl;
    }
    else
    {
        vector<real> results;
        results.resize(options.N);
        run(options, yield, results, args);
        
        if (!args.test)
        {
            Arrays::write_array(cout, results);
        }
    }
}

int main(int argc, char *argv[])
{
    // Args args(argc, argv);
    Args args;
    args.options = "../data/options-60000.in";
    args.yield = "../data/yield.in";
    args.version = 3;
    args.blockSize = 256;
    args.runs = 0;
    args.test = false;
    args.sort = SortType::NONE;

    computeAllOptions(args);

    return 0;
}

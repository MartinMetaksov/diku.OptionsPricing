#include "hip/hip_runtime.h"
#define CUDA
#include "../common/Real.hpp"
#include "../common/OptionConstants.hpp"
#include "../common/FutharkArrays.hpp"
#include "../common/Domain.hpp"
#include "../cuda/CudaErrors.cuh"

#include <chrono>

using namespace std;
using namespace chrono;

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

__global__ void
computeSingleOptionKernel(real *res, OptionConstants *options, real *QsAll, real *QsCopyAll, real *alphasAll, int *QsInd, int *alphasInd)
{
    auto c = options[blockIdx.x];
    auto Qs = QsAll + QsInd[blockIdx.x];
    auto QsCopy = QsCopyAll + QsInd[blockIdx.x];
    auto alphas = alphasAll + alphasInd[blockIdx.x];

    // some test result
    res[blockIdx.x] = c.n;
}

void computeCuda(OptionConstants *options, real *result, int count, bool isTest = false)
{
    // Compute indices
    int* QsInd = new int[count];
    int* alphasInd = new int[count];
    QsInd[0] = 0;
    alphasInd[0] = 0;
    int totalQsCount = 0;
    int totalAlphasCount = 0;
    for (auto i = 0; i < count - 1; ++i)
    {
        auto &option = options[i];
        totalQsCount += option.width;
        totalAlphasCount += option.n + 1;
        QsInd[i + 1] = totalQsCount;
        alphasInd[i + 1] = totalAlphasCount;
    }
    totalQsCount += options[count - 1].width;
    totalAlphasCount += options[count - 1].n + 1;
    
    auto blockSize = 1;

    if (isTest)
    {
        int memorySize = count * sizeof(real) + count * sizeof(OptionConstants) + 2 * count * sizeof(int)
                        + 2 * totalQsCount * sizeof(real) + totalAlphasCount * sizeof(real);
        cout << "Running trinomial option pricing for " << count << " options with block size " << blockSize << endl;
        cout << "Global memory size " << memorySize << endl;
    }

    auto time_begin = steady_clock::now();

    real *d_result, *d_Qs, *d_QsCopy, *d_alphas;
    int *d_QsInd, *d_alphasInd;
    OptionConstants *d_options;
    CudaSafeCall(hipMalloc((void **)&d_result, count * sizeof(real)));
    CudaSafeCall(hipMalloc((void **)&d_options, count * sizeof(OptionConstants)));
    CudaSafeCall(hipMalloc((void **)&d_QsInd, count * sizeof(int)));
    CudaSafeCall(hipMalloc((void **)&d_alphasInd, count * sizeof(int)));
    CudaSafeCall(hipMalloc((void **)&d_Qs, totalQsCount * sizeof(real)));
    CudaSafeCall(hipMalloc((void **)&d_QsCopy, totalQsCount * sizeof(real)));
    CudaSafeCall(hipMalloc((void **)&d_alphas, totalAlphasCount * sizeof(real)));

    hipMemcpy(d_options, options, count * sizeof(OptionConstants), hipMemcpyHostToDevice);
    hipMemcpy(d_QsInd, QsInd, count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_alphasInd, alphasInd, count * sizeof(int), hipMemcpyHostToDevice);

    auto time_begin_kernel = steady_clock::now();
    computeSingleOptionKernel<<<count, blockSize>>>(d_result, d_options, d_Qs, d_QsCopy, d_alphas, d_QsInd, d_alphasInd);
    hipDeviceSynchronize();
    auto time_end_kernel = steady_clock::now();

    CudaCheckError();

    // Copy result
    hipMemcpy(result, d_result, count * sizeof(real), hipMemcpyDeviceToHost);

    hipFree(d_result);
    hipFree(d_options);

    auto time_end = steady_clock::now();
    if (isTest)
    {
        cout << "Kernel executed in " << duration_cast<milliseconds>(time_end_kernel - time_begin_kernel).count() << " ms" << endl;
        cout << "Total GPU time: " << duration_cast<milliseconds>(time_end - time_begin).count() << " ms" << endl
             << endl;
    }
}


void computeAllOptions(const string &filename, bool isTest = false)
{
    // Read options from filename, allocate the result array
    auto options = Option::read_options(filename);
    auto result = new real[options.size()];
    auto optionConstants = new OptionConstants[options.size()];

    for (int i = 0; i < options.size(); ++i)
    {
        optionConstants[i] = OptionConstants::computeConstants(options.at(i));
    }

    computeCuda(optionConstants, result, options.size(), isTest);

    FutharkArrays::write_futhark_array(result, options.size());

    delete[] result;
}

int main(int argc, char *argv[])
{
    bool isTest = false;
    string filename;
    for (int i = 1; i < argc; ++i)
    {
        if (strcmp(argv[i], "-test") == 0)
        {
            isTest = true;
        }
        else
        {
            filename = argv[i];
        }
    }

    computeAllOptions(filename, isTest);

    return 0;
}
